#include "hip/hip_runtime.h"
#include "mandelbrot.h"
#include "cuda_mandelbrot.h"

#include <stdio.h>
#include <stdlib.h>

struct cuda_mandelbrot
{
    uint32_t *v_color_list;
    uint32_t *v_output;
};

__device__
void mb_iteration(float *r_out, float *i_out, float z_r, float z_i, float c_r, float c_i)
{
    *r_out = z_r*z_r - z_i*z_i;
    //*i_out = z_i*z_r + z_r*z_i;
    *i_out = 2*z_r*z_i;

    *r_out += c_r;
    *i_out += c_i;
}

__device__
void mandelbrot_pixel(uint32_t max_iterations, float r, float i, uint32_t *colors, uint32_t *dest)
{
    float z_r = 0;
    float z_i = 0;

    for (int iter = 0; iter < max_iterations; ++iter) {
        mb_iteration(&z_r, &z_i, z_r, z_i, r, i);
        if (z_r * z_r + z_i * z_i > 4.0f) {
            *dest = colors[iter];
            return;
        }
    }

    *dest = 0;
}

__global__
void cu_mandelbrot_kernel(
    int screen_offset_x,
    float zoom,
    float center_x, float center_y,
    float aspect,
    uint32_t max_iterations,
    int w, int h,
    uint32_t *colors,
    uint32_t *output)
{
    /* Given that a maximum of 1024 threads per block can be spawned, we add the screen_offset_x*/
    int x = threadIdx.x + screen_offset_x;
    int y = blockIdx.x;

    /* convert from pixel(x, y) to complex plane(r, i) */
    float r = center_x + ((float)x - (float)w / 2) * zoom / w;
    float i = center_y - ((float)y - (float)h / 2) * zoom * aspect / w;

    mandelbrot_pixel(
        max_iterations,
        r, i,
        colors,
        &output[y * w + x]
    );
}

static void cuda_mandelbrot_step(struct mandelbrot_state *s, int w, int h)
{
    struct cuda_mandelbrot *cuda_ctx = (struct cuda_mandelbrot*)s->accel->__priv;

    for (int off_x = 0; off_x < w; off_x += 1024) {
        int kernel_w = (w - off_x > 1024) ? 1024 : (w - off_x);

        cu_mandelbrot_kernel<<<h, kernel_w>>>(
            off_x,
            s->zoom,
            s->center_x, s->center_y,
            s->aspect, s->max_iters,
            w, h,
            cuda_ctx->v_color_list,
            cuda_ctx->v_output
        );
    }

    hipMemcpy(
        s->output,
        cuda_ctx->v_output,
        sizeof(uint32_t) * w * h,
        hipMemcpyDeviceToHost
    );
}

static void cuda_mandelbrot_destroy(struct mandelbrot_state *s)
{
    struct cuda_mandelbrot *cuda_ctx = (struct cuda_mandelbrot *)s->accel->__priv;
    hipFree(cuda_ctx->v_output);
    hipFree(cuda_ctx->v_color_list);

    free(cuda_ctx);
    free(s->accel);

    s->accel = NULL;
}

struct accelerator_ops cuda_accel_ops {
    .step = cuda_mandelbrot_step,
    .destroy = cuda_mandelbrot_destroy
};

int cuda_init_mandelbrot(struct mandelbrot_state *s, int w, int h)
{
    struct cuda_mandelbrot *cuda_ctx = (struct cuda_mandelbrot*)malloc(
        sizeof(struct cuda_mandelbrot)
    );
    if (!cuda_ctx) {
        goto err_cuda_ctx;
    }

    hipMalloc(&cuda_ctx->v_color_list, sizeof(uint32_t) * k_num_colors);
    if (!cuda_ctx->v_color_list) {
        goto err_cuda_v_color_list;
    }
    hipMalloc(&cuda_ctx->v_output, sizeof(float) * w * h);
    if (!cuda_ctx->v_output) {
        goto err_cuda_v_output;
    }

    hipMemcpy(
        cuda_ctx->v_color_list,
        s->color_list,
        sizeof(uint32_t) * k_num_colors,
        hipMemcpyHostToDevice
    );

    s->accel = (struct accelerator*)malloc(sizeof(struct accelerator));
    if (!s->accel) {
        goto err_accelerator;
    }
    s->accel->__priv = cuda_ctx;
    s->accel->ops = cuda_accel_ops;

    return 0;

err_accelerator:
    hipFree(cuda_ctx->v_output);
err_cuda_v_output:
    hipFree(cuda_ctx->v_color_list);
err_cuda_v_color_list:
    free(cuda_ctx);
err_cuda_ctx:
    return -1;
}
